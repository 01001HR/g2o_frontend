#include "hip/hip_runtime.h"
#include "cudatest.h"
#include <cstdio>
#include <fstream>
using namespace std;

//#define __device__ inline
//#define __host__ inline


__device__ void vecFill(float* v, float x, int n){
  for(int i=0; i<n; i++)
    v[i]=x;
}

__device__ void vecScale(float* v, float s, int n){
  for(int i=0; i<n; i++)
    v[i]*=s;
}

__device__ void vecCopy(float* dest, const float* src, int n){
  for(int i=0; i<n; i++)
    dest[i]=src[i];
}

__device__ void vecSum(float*dest, const float* src, float scale, int n){
  for(int i=0; i<n; i++)
    dest[i]+=scale*src[i];
}

__device__ float vecDot(const float* v1, float* v2, int n){
  float a=0;
  for(int i=0; i<n; i++)
    a+=v1[i]*v2[i];
  return a;
}

__device__ void matVecMul(float* dest, const float* A, const float*b, int rows, int cols){
  vecFill(dest, 0, rows);
  for (int i=0; i<cols; i++){
    vecSum(dest,A,b[i],rows);
    A+=rows;
  }
}

__device__ void matMatMul(float* dest, const float* A, const float*B, int ra, int ca, int cb){
  float* dptr = dest;
  const float* bptr = B;
  for (int i=0; i<cb; i++){
    matVecMul(dptr,A,bptr,ra,ca);
    dptr += ra;
    bptr += ca;
  }
}


__device__ void matTranspose(float* dest, const float* src, int rows, int cols){
  for (int i=0; i<cols; i++)
    for (int j=0; j<rows; j++)
      dest[j*cols+i] = src[i*rows+j];
}


template <int n>
__device__ void vecFill(float* v, float x){
  for(int i=0; i<n; i++)
    v[i]=x;
}

template <int n>
__device__ void vecScale(float* v, float s){
  for(int i=0; i<n; i++)
    v[i]*=s;
}

template <int n>
__device__ void vecCopy(float* dest, const float* src){
  for(int i=0; i<n; i++)
    dest[i]=src[i];
}

template <int n>
__device__ void vecSum(float*dest, const float* src, float scale){
  for(int i=0; i<n; i++)
    dest[i]+=scale*src[i];
}

template <int n>
__device__ float vecDot(const float* v1, const float* v2){
  float a=0;
  for(int i=0; i<n; i++)
    a+=v1[i]*v2[i];
  return a;
}

template <int rows, int cols>
__device__ void matVecMul(float* dest, const float* A, const float*b){
  vecFill<rows>(dest, 0);
  for (int i=0; i<cols; i++){
    vecSum<rows>(dest,A,b[i]);
    A+=rows;
  }
}

template <int ra, int ca, int cb>
__device__ void matMatMul(float* dest, const float* A, const float*B){
  float* dptr = dest;
  const float* bptr = B;
  for (int i=0; i<cb; i++){
    matVecMul<ra,ca>(dptr,A,bptr);
    dptr += ra;
    bptr += ca;
  }
}


template <int rows, int cols>
__device__ void matTranspose(float* dest, const float* src){
  for (int i=0; i<cols; i++)
    for (int j=0; j<rows; j++)
      dest[j*cols+i] = src[i*rows+j];
}

__device__ void matBuildSkew(float* m, const float* v){
  const float x = 2*v[0];
  const float y = 2*v[1];
  const float z = 2*v[2];
  m[0] =  0;   m[4] =  z;  m[8]  = -y; m[12] = 0;   
  m[1] = -z;   m[5] =  0;  m[9]  =  x; m[13] = 0;   
  m[2] =  y;   m[6] = -x;  m[10] =  0; m[14] = 0;   
  m[3] =  0;   m[7] =  0;  m[11] =  0; m[15] = 0;   
}

__device__ void transformInverse(float* d, const float* s) {
  d[0] =  s[0];   d[4] =  s[1];  d[8]  = s[2];  d[12] = 0;   
  d[1] =  s[4];   d[5] =  s[5];  d[9]  = s[6];  d[13] = 0;   
  d[2] =  s[8];   d[6] =  s[9];  d[10] = s[10]; d[14] = 0;   
  d[3] =  0;      d[7] =    0;   d[11] =  0; d[15] = 1;
  float t[4];
  matVecMul<4,4>(t,d,s+12);
  d[12] = -t[0];
  d[13] = -t[1];
  d[14] = -t[2];
}

__device__ void _v2t(float* m, const float* v) {
  const float& tx = v[0];
  const float& ty = v[1];
  const float& tz = v[2];
  const float& qx = v[3];
  const float& qy = v[4];
  const float& qz = v[5];
  const float qw = sqrt(1.f - vecDot<3>(v+3,v+3));
  float _m[] = {qw*qw + qx*qx - qy*qy - qz*qz,  2*(qx*qy + qz*qw),              2*(qx*qz - qy*qw),              0, 
		2*(qx*qy - qw*qz) ,             qw*qw - qx*qx + qy*qy - qz*qz,  2*(qy*qz + qx*qw),               0,
		2*(qx*qz + qw*qy),              2*(qy*qz - qx*qw),              qw*qw - qx*qx - qy*qy + qz*qz,  0,
		tx,                             ty,                             tz,                             1
  };
  vecCopy<16>(m,_m);
}

__device__ void _t2v(float* v, const float* m) {
  const float& m00 = m[0];
  const float& m10 = m[1];
  const float& m20 = m[2];
  //const float& m30 = m[3];
  const float& m01 = m[4];
  const float& m11 = m[5];
  const float& m21 = m[6];
  //const float& m31 = m[7];
  const float& m02 = m[8];
  const float& m12 = m[9];
  const float& m22 = m[10];
  //const float& m32 = m[11];
  v[0]  = m[12];
  v[1]  = m[13];
  v[2]  = m[14];
  float tr = m00 + m11 + m22;
  float qx, qy, qz, qw;
  if (tr > 0) { 
    float S = sqrt(tr+1.0) * 2; // S=4*qw 
    qw = 0.25 * S;
    qx = (m21 - m12) / S;
    qy = (m02 - m20) / S; 
    qz = (m10 - m01) / S; 
  } else if ((m00 > m11)&(m00 > m22)) { 
    float S = sqrt(1.0 + m00 - m11 - m22) * 2; // S=4*qx 
    qw = (m21 - m12) / S;
    qx = 0.25 * S;
    qy = (m01 + m10) / S; 
    qz = (m02 + m20) / S; 
  } else if (m11 > m22) { 
    float S = sqrt(1.0 + m11 - m00 - m22) * 2; // S=4*qy
    qw = (m02 - m20) / S;
    qx = (m01 + m10) / S; 
    qy = 0.25 * S;
    qz = (m12 + m21) / S; 
  } else { 
    float S = sqrt(1.0 + m22 - m00 - m11) * 2; // S=4*qz
    qw = (m10 - m01) / S;
    qx = (m02 + m20) / S;
    qy = (m12 + m21) / S;
    qz = 0.25 * S;
  }
  v[3] = qx;
  v[4] = qy;
  v[5] = qz;
}




__device__ void Aligner_projectPoint(const AlignerContext* context, const float* point){
  float ip[4];
  matVecMul<4,4>(ip, context->KT, point);
  float iw=1./ip[2];
  int x = iw*ip[0];
  int y = iw*ip[1];
  int d = 1000*ip[2];
  int pixelPos =context->referenceDepths.rows*y + x;
  if (d<0 || d > context->maxDepth || pixelPos > context->referenceDepths.rows*context->referenceDepths.cols)
    return;
  atomicMin((context->referenceDepths.values+pixelPos),d);
}



__device__ void IntMatrix_map(IntMatrix* img, int rows, int cols, int* points){
  img->rows=rows;
  img->cols=cols;
  img->values=points;
};

__device__  int IntMatrix_getPointAt(IntMatrix* img, int r, int c) {
  return img->values[c*img->rows+r];
}

__device__ void IntMatrix_setPointAt(IntMatrix* img, int r, int c, int v) {
  img->values[c*img->rows+r]=v;
}



template <int rows>
__device__ void FloatMatrix_map(FloatMatrix<rows>* m, int /*_rows*/, int cols, float* values){
  //m->rows=rows;
  m->cols=cols;
  m->values=values;
}

template <int rows>
__device__ const float* FloatMatrix_getColumnAt(const FloatMatrix<rows>* m, int i){
  return m->values+(i*rows);
}

template <int rows>
__device__ void FloatMatrix_setColumnAt(FloatMatrix<rows>* m, int i, const float* src){
  float* dest = m->values+(i*m->rows);
  vecCopy(dest, src, m->rows);
}


__device__ int Aligner_processCorrespondence(float* error,
					     float* Htt,
					     float* Hrr,
					     float* Htr,
					     float* bt,
					     float* br,
					     int referenceIndex, int currentIndex,
					     const AlignerContext* context,
					     int* noNormal,
					     int* tooDistant,
					     int* badCurvature){

  const float* currentPoint    =  FloatMatrix_getColumnAt(&context->currentPoints, currentIndex);
  const float* currentNormal   =  FloatMatrix_getColumnAt(&context->currentNormals, currentIndex);
  const float* omegaP          =  FloatMatrix_getColumnAt(&context->currentOmegaPs, currentIndex);
  const float* omegaN          =  FloatMatrix_getColumnAt(&context->currentOmegaNs, currentIndex);
  float   currentCurvature     =  context->currentCurvatures[currentIndex];
  const float* referencePoint_  =  FloatMatrix_getColumnAt(&context->referencePoints, referenceIndex);
  const float* referenceNormal_ =  FloatMatrix_getColumnAt(&context->referenceNormals, referenceIndex);
  float referenceCurvature     =  context->referenceCurvatures[referenceIndex];
  


  float referencePoint[4];
  float referenceNormal[4];
  float pointsDifference[4];

  const float* T=context->transform;


  // float _trp[4], _trn[4];
  // vecCopy<4>(_trp,referencePoint_);
  // vecCopy<4>(_trn,referenceNormal_);
  //_trp[3]=1.0f;
  //_trn[3]=0.0f;


  matVecMul<4,4>(referencePoint,T,referencePoint_);
  matVecMul<4,4>(referenceNormal,T,referenceNormal_);

  referenceCurvature = (referenceCurvature<context->flatCurvatureThreshold)?context->flatCurvatureThreshold:referenceCurvature;

  currentCurvature = (currentCurvature<context->flatCurvatureThreshold)?context->flatCurvatureThreshold:currentCurvature; 
  float curvatureRatio=(referenceCurvature + 1e-5)/(currentCurvature + 1e-5);
  float normalsRatio = vecDot<4>(currentNormal,referenceNormal);
  
  vecCopy<4>(pointsDifference,referencePoint);
  vecSum<4>(pointsDifference,currentPoint,-1.0f);
  float pointsDistance=vecDot<4>(pointsDifference,pointsDifference);

  if (normalsRatio < context->normalThreshold){
    (*noNormal)++;
    return 0;
  }
  if (pointsDistance > context->distanceThreshold){
    (*tooDistant)++;
    return 0;
  }
  if ((curvatureRatio < context->minCurvatureRatio) ||
      (curvatureRatio > context->maxCurvatureRatio)){
    (*badCurvature)++;
    return 0;
  }
    
    

  int increment = 
    (normalsRatio > context->normalThreshold) &
    (pointsDistance < context->distanceThreshold) &
    (curvatureRatio > context->minCurvatureRatio) &
    (curvatureRatio < context->maxCurvatureRatio);
  if (! increment)  
    return 0;
  
  float normalsDifference[4];
  vecCopy<4>(normalsDifference,referenceNormal);
  vecSum<4>(normalsDifference,currentNormal,-1.0f);
  

  //const Vector4f ep = omegaP*pointError;
  float ep[4];
  matVecMul<4,4>(ep, omegaP, pointsDifference);

  //const Vector4f en = omegaN*normalError;
  float en[4];
  matVecMul<4,4>(en, omegaN, normalsDifference);

  //matPrint(ep,4,1,"ep");
  //matPrint(en,4,1,"en");

  //float localError = pointError.dot(ep) + normalError.dot(en);
  
  float localError = vecDot<4>(ep,pointsDifference) + vecDot<4>(en,normalsDifference);

  int chiOk = localError < context->inlierThreshold;
  //if (! chiOk)     return 0;
  float scale = chiOk * increment; // scale is = 0 if we are in front of an inlier

  
  //Matrix4f Sp = skew(referencePoint);
  float Sp[16];
  matBuildSkew(Sp,referencePoint);
  // compute the transposed (for the skew is mult by -1)

  //Matrix4f Sn = skew(referenceNormal);
  float Sn[16];
  matBuildSkew(Sn,referenceNormal);


  //Htt = omegaP;
  vecCopy<16>(Htt,omegaP);
  
  // prepare to undo if necessary
  vecScale<16>(Htt,scale);

  //Htr.noalias() = omegaP*Sp;
  matMatMul<4,4,4>(Htr,omegaP,Sp);

  //Hrr.noalias() = - (Sp*omegaP*Sp + Sn*omegaN*Sn);
  float temp[16], temp2[16];
  matMatMul<4,4,4>(Hrr,Sp,Htr);
  matMatMul<4,4,4>(temp,Sn,omegaN);
  matMatMul<4,4,4>(temp2,temp,Sn);
  vecSum<16>(Hrr,temp2,+1.0f);
  vecScale<16>(Hrr,-1.0f);
  //bt.noalias() = ep;

  vecCopy<4>(bt,ep);


  //br.noalias() = - (Sp*ep + Sn*en);
  matVecMul<4,4>(br,Sp,ep);
  matVecMul<4,4>(temp,Sn,en);
  vecSum<4>(br,temp,+1.0f);
  vecScale<4>(br,-1.0f);

  // vecScale<4>(bt,scale);
  // vecScale<4>(br,scale);
  // vecScale<4>(Htt,scale);
  // vecScale<4>(Htr,scale);
  // vecScale<4>(Hrr,scale);

  *error = localError;
  return scale*chiOk;
}


#ifdef REAL_CUDA_THING
// src is the base of the shared data of the kernel;
__device__ void ReduceMatrix_shared(float* blockResult, int tid, int bdim, int bidx){
  // contiguous range pattern
  extern __shared__ float sdata[];
  for(int offset = bdim / 2;  offset > 0; offset >>= 1)
    {
      if(tid < offset)
	{
	  // add a partial sum upstream to our own
	  vecSum<56>(sdata +(tid * 56), sdata +( (tid + offset) * 56), +1);
	}
      
      // wait until all threads in the block have
      // updated their partial sums
      __syncthreads();
    }
  if (tid == 0)
    vecCopy<56>(blockResult + (bidx*56),sdata);
  __syncthreads();
}

__global__ void ReduceMatrix_global(float* blockResult, float* input, int n){
  extern __shared__ float sdata[];
  //unsigned int i = bdim * bidx + tid;
  float x[56];
  vecFill<56>(x,0);
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<n){
    vecCopy<56>(x,input+(i*56));
  }
  vecCopy<56>(sdata+(threadIdx.x*56),x);
  ReduceMatrix_shared(blockResult, threadIdx.x, blockDim.x, blockIdx.x);
}


__global__ void Aligner_1stKernel(AlignerContext* context){
  extern __shared__ float sdata[];
  //unsigned int i = bdim * bidx + tid;
  float x[56];
  vecFill<56>(x,0);
  int i = blockDim.x * blockIdx.x + threadIdx.x;
 
  int retval = 0;
  float error;
  float Htt[16];
  float Hrr[16];
  float Htr[16];
  float bt[4];
  float br[4];
  if (i < context->currentIndices.rows*context->currentIndices.cols) {
    vecCopy<56>(sdata+(threadIdx.x*56),x);
    int referenceIndex=context->referenceIndices.values[i];
    int currentIndex=context->currentIndices.values[i];
    int noNormal, tooDistant, badCurvature;
    retval = Aligner_processCorrespondence(&error, Htt, Hrr, Htr,
					   bt, br,
					   referenceIndex, 
					   currentIndex, context,
					   &noNormal, &tooDistant, &badCurvature);
  }
  __syncthreads();
  float* dest = sdata + threadIdx.x*56;
  if (retval){
    vecCopy<16>(dest,Htt);
    vecCopy<16>(dest+16,Htr);
    vecCopy<16>(dest+32,Hrr);
    vecCopy<4>(dest+48,bt);
    vecCopy<4>(dest+52,br);
  }
  __syncthreads();
  ReduceMatrix_shared(context->tempsums, threadIdx.x, blockDim.x, blockIdx.x);
};


void Aligner_doCudaStuff(AlignerContext* aligner){
  int numElements = aligner->referenceIndices.rows * aligner->referenceIndices.cols;
  int threadsPerBlock = 256;
  int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  Aligner_1stKernel<<<blocksPerGrid,threadsPerBlock>>>(aligner->cudaAligner);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to launch the 1st kernel\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  while (blocksPerGrid>0){
    int oldBlocksPerGrid=blocksPerGrid;
    blocksPerGrid=blocksPerGrid/threadsPerBlock;
    ReduceMatrix_global<<<blocksPerGrid,threadsPerBlock>>>(aligner->cudaAligner->tempsums, 
							   aligner->cudaAligner->tempsums,
							   oldBlocksPerGrid);

    err = hipGetLastError();
    if (err != hipSuccess)
      {
	fprintf(stderr, "Failed to launch the reduction kernel!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
      }
  }
}


#endif

#if 0

void Aligner::align() {
  _projector->setTransform(Isometry3f::Identity());
  _projector->project(_currentIndexImage,
		      _currentDepthImage,
		      *_currentPoints);
  _T = _initialGuess;
  //Vector6f myFuckingT;   myFuckingT << 0.1, 0.01, 0.03, 0, 0, 0;   _T=v2t(myFuckingT);
  // fill the cuda context
  float referenceCurvatures[_referencePoints->size()];
  for (size_t i=0; i<_referencePoints->size(); i++)
    referenceCurvatures[i] = _referenceStats->at(i).curvature();
  
  float currentCurvatures[_currentPoints->size()];
  for (size_t i=0; i<_currentPoints->size(); i++)
    currentCurvatures[i] = _currentStats->at(i).curvature();


  for(int i = 0; i < _outerIterations; i++) {
    cout << "********************* Iteration " << i << " *********************" << endl;
    
    /************************************************************************
     *                         Correspondence Computation                   *
     ************************************************************************/
    cout << "Computing correspondences...";
    
    _projector->setTransform(_T.inverse());
    _projector->project(_referenceIndexImage,
			_referenceDepthImage,
			*_referencePoints);
    
    double tCorrStart = g2o::get_time();
    // Correspondences computation.    
    _correspondenceGenerator.compute(_correspondences,
				     *_referencePoints, *_currentPoints,
				     *_referenceNormals, *_currentNormals,
				     _referenceIndexImage, _currentIndexImage,
				     *_referenceStats, *_currentStats,
				     _T);
    double tCorrEnd = g2o::get_time();
    
    cout << " done." << endl;
    _numCorrespondences = _correspondenceGenerator.numCorrespondences();
    cout << "# inliers found: " << _numCorrespondences << endl;
 
    
    {
      _T.matrix().block<1, 4>(3, 0) << 0, 0, 0, 1;
      
      Aligner_fillContext(&cudaContext,
			  &(_referencePoints->at(0).coeffRef(0,0)), 
			  &(_referenceNormals->at(0).coeffRef(0,0)), 
			  referenceCurvatures,
			  _referencePoints->size(),
			  &(_currentPoints->at(0).coeffRef(0,0)), 
			  &(_currentNormals->at(0).coeffRef(0,0)), 
			  currentCurvatures,
			  &(_currentPointOmegas->at(0).coeffRef(0,0)), 
			  &(_currentNormalOmegas->at(0).coeffRef(0,0)), 
			  _currentPoints->size(),
			  &(_referenceIndexImage.coeffRef(0,0)),
			  &(_currentIndexImage.coeffRef(0,0)),
			  _referenceIndexImage.rows(),
			  _referenceIndexImage.cols(),
			  &_T.matrix().coeffRef(0,0));
      
      cerr << "T: " << _T.matrix() << endl;
      cudaContext.distanceThreshold = 
	_correspondenceGenerator.inlierDistanceThreshold() * 
	_correspondenceGenerator.inlierDistanceThreshold() ;
      cudaContext.flatCurvatureThreshold = _correspondenceGenerator.flatCurvatureThreshold();
      cudaContext.normalThreshold = _correspondenceGenerator.inlierNormalAngularThreshold();
      cudaContext.minCurvatureRatio = 1./_correspondenceGenerator.inlierCurvatureRatioThreshold();
      cudaContext.maxCurvatureRatio = _correspondenceGenerator.inlierCurvatureRatioThreshold();
      cudaContext.inlierThreshold = _linearizer->inlierMaxChi2();
      
    }
    /************************************************************************
     *                            Alignment                                 *
     ************************************************************************/
    for (int k = 0; k < _innerIterations; k++) {      
      Matrix6f H;
      Vector6f b;
      _T.matrix().block<1, 4>(3, 0) << 0, 0, 0, 1;
      double tLinStart = g2o::get_time();
      _linearizer->setT(_T);
      _linearizer->update();
      double tLinEnd = g2o::get_time();
      cerr << "inliers after computing the error" << _linearizer->inliers() << endl;
      //H = _linearizer->H() + Matrix6f::Identity();
      //b = _linearizer->b();

      

      Matrix6f Hgt;
      Vector6f bgt;

      double tStart = g2o::get_time();
      float error;
      float 
	Htt[16], 
	Htr[16], 
	Hrr[16], 
	bt[4], 
	br[4];
      int inliers = Aligner_processCorrespondences(&error, Htt, Htr, Hrr, bt, br, &cudaContext);
      double tEnd = g2o::get_time();
      cerr << "inliers found by the pre-cuda thing:" << inliers << endl;
      

      cerr << "tcorr: " << tCorrEnd - tCorrStart << endl;
      cerr << "tLin: " << tLinEnd - tLinStart << endl;
      cerr << "tMine: " << tEnd - tStart << endl;

      Eigen::Map<Eigen::Matrix4f> _Htt(Htt);
      Eigen::Map<Eigen::Matrix4f> _Htr(Htr);
      Eigen::Map<Eigen::Matrix4f> _Hrr(Hrr);
      Eigen::Map<Eigen::Vector4f> _bt(bt);
      Eigen::Map<Eigen::Vector4f> _br(br);
 
      Hgt.block<3,3>(0,0) = _Htt.block<3,3>(0,0);
      Hgt.block<3,3>(0,3) = _Htr.block<3,3>(0,0);
      Hgt.block<3,3>(3,3) = _Hrr.block<3,3>(0,0);
      Hgt.block<3,3>(3,0) =  Hgt.block<3,3>(0,3).transpose();
      bgt.block<3,1>(0,0) = _bt.block<3,1>(0,0);
      bgt.block<3,1>(3,0) = _br.block<3,1>(0,0);


      cerr << "H difference: " << ((H-Hgt)*1./H.norm()) << endl;
      cerr << "b difference: " << ((b-bgt)*1./b.norm()) << endl;

      H = Hgt;
      b = bgt;

      H+= Matrix6f::Identity() * 10.0f;
      
      Vector6f dx = H.ldlt().solve(-b);
      Eigen::Isometry3f dT = v2t(dx);
      _T = dT * _T;
      _T.matrix().block<1, 4>(3, 0) << 0, 0, 0, 1;
    }    
  }
  _T = _sensorOffset * _T;
}


#endif



/* simple vet test thing */
/*
int main() {
  float v1[4];
  float v2[4];
  for (int i=0; i<4; i++){
    v1[i]=i;
  }
  matPrint(v1,4,1, "v1");
  vecFill(v2, 2, 4);
  matPrint(v2,4,1, "v2");
  vecScale(v1,-1,4);
  matPrint(v1,4,1, "v1=v1*-1");
  printf("v1*v1 = %f\n", vecDot(v1,v1,4));
  vecSum(v1,v2,2.0,4);
  matPrint(v1,4,1, "v1+2*v2");
  vecCopy(v2,v1,4);
  matPrint(v2,4,1, "v2=v1");
  vecScale(v2,0.1,4);
  matPrint(v2,4,1, "v2*.1");
  float m[16];
  for (int i=0; i<4; i++)
    for (int j=0; j<4; j++)
      m[i*4+j]=((i+1)*(j+1))/16.0f;
  matPrint(m,4,4,"m");
  float v3[4];
  matVecMul(v3,m,v2,4,4);
  matPrint(v3,4,1,"m*v2");
  float m3[16];
  matMatMul(m3, m, m, 4,4,4);
  matPrint(m3,4,4,"m*m");
  matBuildSkew(m3,v1);
  matPrint(m3,4,4,"skew(v1)");
  float vm[4];
  matMatMul(vm,m3,v1,4,4,1);
  matPrint(vm,4,1,"non square matrix product");
  
}
*/
