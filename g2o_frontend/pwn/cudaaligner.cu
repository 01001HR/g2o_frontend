#include "hip/hip_runtime.h"
#include "cudatest.h"
#include <cstdio>
#include <fstream>
using namespace std;

#include "cudasla.cuh"

//#define __device__ inline
//#define __host__ inline

__device__ void Aligner_projectPoint(const AlignerContext* context, const float* point){
  float ip[4];
  matVecMul<4,4>(ip, context->KT, point);
  float iw=1./ip[2];
  int x = iw*ip[0];
  int y = iw*ip[1];
  int d = 1000*ip[2];
  int pixelPos =context->referenceDepths.rows*y + x;
  if (d<0 || d > context->maxDepth || pixelPos > context->referenceDepths.rows*context->referenceDepths.cols)
    return;
  atomicMin((context->referenceDepths.values+pixelPos),d);
}



__device__ void IntMatrix_map(IntMatrix* img, int rows, int cols, int* points){
  img->rows=rows;
  img->cols=cols;
  img->values=points;
};

__device__  int IntMatrix_getPointAt(IntMatrix* img, int r, int c) {
  return img->values[c*img->rows+r];
}

__device__ void IntMatrix_setPointAt(IntMatrix* img, int r, int c, int v) {
  img->values[c*img->rows+r]=v;
}



template <int rows>
__device__ void FloatMatrix_map(FloatMatrix<rows>* m, int /*_rows*/, int cols, float* values){
  //m->rows=rows;
  m->cols=cols;
  m->values=values;
}

template <int rows>
__device__ const float* FloatMatrix_getColumnAt(const FloatMatrix<rows>* m, int i){
  return m->values+(i*rows);
}

template <int rows>
__device__ void FloatMatrix_setColumnAt(FloatMatrix<rows>* m, int i, const float* src){
  float* dest = m->values+(i*m->rows);
  vecCopy(dest, src, m->rows);
}


__device__ int Aligner_processCorrespondence(float* error,
					     float* Htt,
					     float* Hrr,
					     float* Htr,
					     float* bt,
					     float* br,
					     int referenceIndex, int currentIndex,
					     const AlignerContext* context,
					     int* noNormal,
					     int* tooDistant,
					     int* badCurvature){

  const float* currentPoint    =  FloatMatrix_getColumnAt(&context->currentPoints, currentIndex);
  const float* currentNormal   =  FloatMatrix_getColumnAt(&context->currentNormals, currentIndex);
  const float* omegaP          =  FloatMatrix_getColumnAt(&context->currentOmegaPs, currentIndex);
  const float* omegaN          =  FloatMatrix_getColumnAt(&context->currentOmegaNs, currentIndex);
  float   currentCurvature     =  context->currentCurvatures[currentIndex];
  const float* referencePoint_  =  FloatMatrix_getColumnAt(&context->referencePoints, referenceIndex);
  const float* referenceNormal_ =  FloatMatrix_getColumnAt(&context->referenceNormals, referenceIndex);
  float referenceCurvature     =  context->referenceCurvatures[referenceIndex];
  


  float referencePoint[4];
  float referenceNormal[4];
  float pointsDifference[4];

  const float* T=context->transform;


  // float _trp[4], _trn[4];
  // vecCopy<4>(_trp,referencePoint_);
  // vecCopy<4>(_trn,referenceNormal_);
  //_trp[3]=1.0f;
  //_trn[3]=0.0f;


  matVecMul<4,4>(referencePoint,T,referencePoint_);
  matVecMul<4,4>(referenceNormal,T,referenceNormal_);

  referenceCurvature = (referenceCurvature<context->flatCurvatureThreshold)?context->flatCurvatureThreshold:referenceCurvature;

  currentCurvature = (currentCurvature<context->flatCurvatureThreshold)?context->flatCurvatureThreshold:currentCurvature; 
  float curvatureRatio=(referenceCurvature + 1e-5)/(currentCurvature + 1e-5);
  float normalsRatio = vecDot<4>(currentNormal,referenceNormal);
  
  vecCopy<4>(pointsDifference,referencePoint);
  vecSum<4>(pointsDifference,currentPoint,-1.0f);
  float pointsDistance=vecDot<4>(pointsDifference,pointsDifference);

  if (normalsRatio < context->normalThreshold){
    (*noNormal)++;
    return 0;
  }
  if (pointsDistance > context->distanceThreshold){
    (*tooDistant)++;
    return 0;
  }
  if ((curvatureRatio < context->minCurvatureRatio) ||
      (curvatureRatio > context->maxCurvatureRatio)){
    (*badCurvature)++;
    return 0;
  }
    
    

  int increment = 
    (normalsRatio > context->normalThreshold) &
    (pointsDistance < context->distanceThreshold) &
    (curvatureRatio > context->minCurvatureRatio) &
    (curvatureRatio < context->maxCurvatureRatio);
  if (! increment)  
    return 0;
  
  float normalsDifference[4];
  vecCopy<4>(normalsDifference,referenceNormal);
  vecSum<4>(normalsDifference,currentNormal,-1.0f);
  

  //const Vector4f ep = omegaP*pointError;
  float ep[4];
  matVecMul<4,4>(ep, omegaP, pointsDifference);

  //const Vector4f en = omegaN*normalError;
  float en[4];
  matVecMul<4,4>(en, omegaN, normalsDifference);

  //matPrint(ep,4,1,"ep");
  //matPrint(en,4,1,"en");

  //float localError = pointError.dot(ep) + normalError.dot(en);
  
  float localError = vecDot<4>(ep,pointsDifference) + vecDot<4>(en,normalsDifference);

  int chiOk = localError < context->inlierThreshold;
  //if (! chiOk)     return 0;
  float scale = chiOk * increment; // scale is = 0 if we are in front of an inlier

  
  //Matrix4f Sp = skew(referencePoint);
  float Sp[16];
  matBuildSkew(Sp,referencePoint);
  // compute the transposed (for the skew is mult by -1)

  //Matrix4f Sn = skew(referenceNormal);
  float Sn[16];
  matBuildSkew(Sn,referenceNormal);


  //Htt = omegaP;
  vecCopy<16>(Htt,omegaP);
  
  // prepare to undo if necessary
  vecScale<16>(Htt,scale);

  //Htr.noalias() = omegaP*Sp;
  matMatMul<4,4,4>(Htr,omegaP,Sp);

  //Hrr.noalias() = - (Sp*omegaP*Sp + Sn*omegaN*Sn);
  float temp[16], temp2[16];
  matMatMul<4,4,4>(Hrr,Sp,Htr);
  matMatMul<4,4,4>(temp,Sn,omegaN);
  matMatMul<4,4,4>(temp2,temp,Sn);
  vecSum<16>(Hrr,temp2,+1.0f);
  vecScale<16>(Hrr,-1.0f);
  //bt.noalias() = ep;

  vecCopy<4>(bt,ep);


  //br.noalias() = - (Sp*ep + Sn*en);
  matVecMul<4,4>(br,Sp,ep);
  matVecMul<4,4>(temp,Sn,en);
  vecSum<4>(br,temp,+1.0f);
  vecScale<4>(br,-1.0f);

  // vecScale<4>(bt,scale);
  // vecScale<4>(br,scale);
  // vecScale<4>(Htt,scale);
  // vecScale<4>(Htr,scale);
  // vecScale<4>(Hrr,scale);

  *error = localError;
  return scale*chiOk;
}


#ifdef REAL_CUDA_THING
// src is the base of the shared data of the kernel;
__device__ void ReduceMatrix_shared(float* blockResult, int tid, int bdim, int bidx){
  // contiguous range pattern
  extern __shared__ float sdata[];
  for(int offset = bdim / 2;  offset > 0; offset >>= 1)
    {
      if(tid < offset)
	{
	  // add a partial sum upstream to our own
	  vecSum<56>(sdata +(tid * 56), sdata +( (tid + offset) * 56), +1);
	}
      
      // wait until all threads in the block have
      // updated their partial sums
      __syncthreads();
    }
  if (tid == 0)
    vecCopy<56>(blockResult + (bidx*56),sdata);
  __syncthreads();
}

__global__ void ReduceMatrix_global(float* blockResult, float* input, int n){
  extern __shared__ float sdata[];
  //unsigned int i = bdim * bidx + tid;
  float x[56];
  vecFill<56>(x,0);
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<n){
    vecCopy<56>(x,input+(i*56));
  }
  vecCopy<56>(sdata+(threadIdx.x*56),x);
  ReduceMatrix_shared(blockResult, threadIdx.x, blockDim.x, blockIdx.x);
}


__global__ void Aligner_1stKernel(AlignerContext* context){
  extern __shared__ float sdata[];
  //unsigned int i = bdim * bidx + tid;
  float x[56];
  vecFill<56>(x,0);
  int i = blockDim.x * blockIdx.x + threadIdx.x;
 
  int retval = 0;
  float error;
  float Htt[16];
  float Hrr[16];
  float Htr[16];
  float bt[4];
  float br[4];
  if (i < context->currentIndices.rows*context->currentIndices.cols) {
    vecCopy<56>(sdata+(threadIdx.x*56),x);
    int referenceIndex=context->referenceIndices.values[i];
    int currentIndex=context->currentIndices.values[i];
    int noNormal, tooDistant, badCurvature;
    retval = Aligner_processCorrespondence(&error, Htt, Hrr, Htr,
					   bt, br,
					   referenceIndex, 
					   currentIndex, context,
					   &noNormal, &tooDistant, &badCurvature);
  }
  __syncthreads();
  float* dest = sdata + threadIdx.x*56;
  if (retval){
    vecCopy<16>(dest,Htt);
    vecCopy<16>(dest+16,Htr);
    vecCopy<16>(dest+32,Hrr);
    vecCopy<4>(dest+48,bt);
    vecCopy<4>(dest+52,br);
  }
  __syncthreads();
  ReduceMatrix_shared(context->tempsums, threadIdx.x, blockDim.x, blockIdx.x);
};


void Aligner_doCudaStuff(AlignerContext* aligner){
  int numElements = aligner->referenceIndices.rows * aligner->referenceIndices.cols;
  int threadsPerBlock = 256;
  int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
  Aligner_1stKernel<<<blocksPerGrid,threadsPerBlock>>>(aligner->cudaAligner);
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr, "Failed to launch the 1st kernel\n", hipGetErrorString(err));
      exit(EXIT_FAILURE);
    }

  while (blocksPerGrid>0){
    int oldBlocksPerGrid=blocksPerGrid;
    blocksPerGrid=blocksPerGrid/threadsPerBlock;
    ReduceMatrix_global<<<blocksPerGrid,threadsPerBlock>>>(aligner->cudaAligner->tempsums, 
							   aligner->cudaAligner->tempsums,
							   oldBlocksPerGrid);

    err = hipGetLastError();
    if (err != hipSuccess)
      {
	fprintf(stderr, "Failed to launch the reduction kernel!\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
      }
  }
}


#endif

#if 0

void Aligner::align() {
  _projector->setTransform(Isometry3f::Identity());
  _projector->project(_currentIndexImage,
		      _currentDepthImage,
		      *_currentPoints);
  _T = _initialGuess;
  //Vector6f myFuckingT;   myFuckingT << 0.1, 0.01, 0.03, 0, 0, 0;   _T=v2t(myFuckingT);
  // fill the cuda context
  float referenceCurvatures[_referencePoints->size()];
  for (size_t i=0; i<_referencePoints->size(); i++)
    referenceCurvatures[i] = _referenceStats->at(i).curvature();
  
  float currentCurvatures[_currentPoints->size()];
  for (size_t i=0; i<_currentPoints->size(); i++)
    currentCurvatures[i] = _currentStats->at(i).curvature();


  for(int i = 0; i < _outerIterations; i++) {
    cout << "********************* Iteration " << i << " *********************" << endl;
    
    /************************************************************************
     *                         Correspondence Computation                   *
     ************************************************************************/
    cout << "Computing correspondences...";
    
    _projector->setTransform(_T.inverse());
    _projector->project(_referenceIndexImage,
			_referenceDepthImage,
			*_referencePoints);
    
    double tCorrStart = g2o::get_time();
    // Correspondences computation.    
    _correspondenceGenerator.compute(_correspondences,
				     *_referencePoints, *_currentPoints,
				     *_referenceNormals, *_currentNormals,
				     _referenceIndexImage, _currentIndexImage,
				     *_referenceStats, *_currentStats,
				     _T);
    double tCorrEnd = g2o::get_time();
    
    cout << " done." << endl;
    _numCorrespondences = _correspondenceGenerator.numCorrespondences();
    cout << "# inliers found: " << _numCorrespondences << endl;
 
    
    {
      _T.matrix().block<1, 4>(3, 0) << 0, 0, 0, 1;
      
      Aligner_fillContext(&cudaContext,
			  &(_referencePoints->at(0).coeffRef(0,0)), 
			  &(_referenceNormals->at(0).coeffRef(0,0)), 
			  referenceCurvatures,
			  _referencePoints->size(),
			  &(_currentPoints->at(0).coeffRef(0,0)), 
			  &(_currentNormals->at(0).coeffRef(0,0)), 
			  currentCurvatures,
			  &(_currentPointOmegas->at(0).coeffRef(0,0)), 
			  &(_currentNormalOmegas->at(0).coeffRef(0,0)), 
			  _currentPoints->size(),
			  &(_referenceIndexImage.coeffRef(0,0)),
			  &(_currentIndexImage.coeffRef(0,0)),
			  _referenceIndexImage.rows(),
			  _referenceIndexImage.cols(),
			  &_T.matrix().coeffRef(0,0));
      
      cerr << "T: " << _T.matrix() << endl;
      cudaContext.distanceThreshold = 
	_correspondenceGenerator.inlierDistanceThreshold() * 
	_correspondenceGenerator.inlierDistanceThreshold() ;
      cudaContext.flatCurvatureThreshold = _correspondenceGenerator.flatCurvatureThreshold();
      cudaContext.normalThreshold = _correspondenceGenerator.inlierNormalAngularThreshold();
      cudaContext.minCurvatureRatio = 1./_correspondenceGenerator.inlierCurvatureRatioThreshold();
      cudaContext.maxCurvatureRatio = _correspondenceGenerator.inlierCurvatureRatioThreshold();
      cudaContext.inlierThreshold = _linearizer->inlierMaxChi2();
      
    }
    /************************************************************************
     *                            Alignment                                 *
     ************************************************************************/
    for (int k = 0; k < _innerIterations; k++) {      
      Matrix6f H;
      Vector6f b;
      _T.matrix().block<1, 4>(3, 0) << 0, 0, 0, 1;
      double tLinStart = g2o::get_time();
      _linearizer->setT(_T);
      _linearizer->update();
      double tLinEnd = g2o::get_time();
      cerr << "inliers after computing the error" << _linearizer->inliers() << endl;
      //H = _linearizer->H() + Matrix6f::Identity();
      //b = _linearizer->b();

      

      Matrix6f Hgt;
      Vector6f bgt;

      double tStart = g2o::get_time();
      float error;
      float 
	Htt[16], 
	Htr[16], 
	Hrr[16], 
	bt[4], 
	br[4];
      int inliers = Aligner_processCorrespondences(&error, Htt, Htr, Hrr, bt, br, &cudaContext);
      double tEnd = g2o::get_time();
      cerr << "inliers found by the pre-cuda thing:" << inliers << endl;
      

      cerr << "tcorr: " << tCorrEnd - tCorrStart << endl;
      cerr << "tLin: " << tLinEnd - tLinStart << endl;
      cerr << "tMine: " << tEnd - tStart << endl;

      Eigen::Map<Eigen::Matrix4f> _Htt(Htt);
      Eigen::Map<Eigen::Matrix4f> _Htr(Htr);
      Eigen::Map<Eigen::Matrix4f> _Hrr(Hrr);
      Eigen::Map<Eigen::Vector4f> _bt(bt);
      Eigen::Map<Eigen::Vector4f> _br(br);
 
      Hgt.block<3,3>(0,0) = _Htt.block<3,3>(0,0);
      Hgt.block<3,3>(0,3) = _Htr.block<3,3>(0,0);
      Hgt.block<3,3>(3,3) = _Hrr.block<3,3>(0,0);
      Hgt.block<3,3>(3,0) =  Hgt.block<3,3>(0,3).transpose();
      bgt.block<3,1>(0,0) = _bt.block<3,1>(0,0);
      bgt.block<3,1>(3,0) = _br.block<3,1>(0,0);


      cerr << "H difference: " << ((H-Hgt)*1./H.norm()) << endl;
      cerr << "b difference: " << ((b-bgt)*1./b.norm()) << endl;

      H = Hgt;
      b = bgt;

      H+= Matrix6f::Identity() * 10.0f;
      
      Vector6f dx = H.ldlt().solve(-b);
      Eigen::Isometry3f dT = v2t(dx);
      _T = dT * _T;
      _T.matrix().block<1, 4>(3, 0) << 0, 0, 0, 1;
    }    
  }
  _T = _sensorOffset * _T;
}


#endif



/* simple vet test thing */
/*
int main() {
  float v1[4];
  float v2[4];
  for (int i=0; i<4; i++){
    v1[i]=i;
  }
  matPrint(v1,4,1, "v1");
  vecFill(v2, 2, 4);
  matPrint(v2,4,1, "v2");
  vecScale(v1,-1,4);
  matPrint(v1,4,1, "v1=v1*-1");
  printf("v1*v1 = %f\n", vecDot(v1,v1,4));
  vecSum(v1,v2,2.0,4);
  matPrint(v1,4,1, "v1+2*v2");
  vecCopy(v2,v1,4);
  matPrint(v2,4,1, "v2=v1");
  vecScale(v2,0.1,4);
  matPrint(v2,4,1, "v2*.1");
  float m[16];
  for (int i=0; i<4; i++)
    for (int j=0; j<4; j++)
      m[i*4+j]=((i+1)*(j+1))/16.0f;
  matPrint(m,4,4,"m");
  float v3[4];
  matVecMul(v3,m,v2,4,4);
  matPrint(v3,4,1,"m*v2");
  float m3[16];
  matMatMul(m3, m, m, 4,4,4);
  matPrint(m3,4,4,"m*m");
  matBuildSkew(m3,v1);
  matPrint(m3,4,4,"skew(v1)");
  float vm[4];
  matMatMul(vm,m3,v1,4,4,1);
  matPrint(vm,4,1,"non square matrix product");
  
}
*/
